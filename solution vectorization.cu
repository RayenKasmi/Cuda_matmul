#include "hip/hip_runtime.h"
#pragma once

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;
typedef unsigned int uint;

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void sgemmVectorize(int N, float* A, float* B, float* C) {
	const uint cRow = blockIdx.y;
	const uint cCol = blockIdx.x;

	// BN/TN are the number of threads to span a column
	const int threadCol = threadIdx.x % (BN / TN);
	const int threadRow = threadIdx.x / (BN / TN);

	// allocate space for the current blocktile in smem
	__shared__ float As[BM * BK];
	__shared__ float Bs[BK * BN];

	// Move blocktile to beginning of A's row and B's column
	A += cRow * BM * N;
	B += cCol * BN;
	C += cRow * BM * N + cCol * BN;

	// calculating the indices that this thread will load into SMEM
	// we'll load 128bit / 32bit = 4 elements per thread at each step
	const uint innerRowA = threadIdx.x / (BK / 4);
	const uint innerColA = threadIdx.x % (BK / 4);
	const uint innerRowB = threadIdx.x / (BN / 4);
	const uint innerColB = threadIdx.x % (BN / 4);

	// allocate thread-local cache for results in registerfile
	float threadResults[TM * TN] = { 0.0 };
	float regM[TM] = { 0.0 };
	float regN[TN] = { 0.0 };

	// outer-most loop over block tiles
	for (uint bkIdx = 0; bkIdx < N; bkIdx += BK) {
		// populate the SMEM caches
		// transpose A while loading it
		float4 tmp =
			reinterpret_cast<float4*>(&A[innerRowA * N + innerColA * 4])[0];
		As[(innerColA * 4 + 0) * BM + innerRowA] = tmp.x;
		As[(innerColA * 4 + 1) * BM + innerRowA] = tmp.y;
		As[(innerColA * 4 + 2) * BM + innerRowA] = tmp.z;
		As[(innerColA * 4 + 3) * BM + innerRowA] = tmp.w;

		reinterpret_cast<float4*>(&Bs[innerRowB * BN + innerColB * 4])[0] =
			reinterpret_cast<float4*>(&B[innerRowB * N + innerColB * 4])[0];
		__syncthreads();

		// advance blocktile
		A += BK;     // move BK columns to right
		B += BK * N; // move BK rows down

		// calculate per-thread results
		for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
			// block into registers
			for (uint i = 0; i < TM; ++i) {
				regM[i] = As[dotIdx * BM + threadRow * TM + i];
			}
			for (uint i = 0; i < TN; ++i) {
				regN[i] = Bs[dotIdx * BN + threadCol * TN + i];
			}
			for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
				for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
					threadResults[resIdxM * TN + resIdxN] +=
						regM[resIdxM] * regN[resIdxN];
				}
			}
		}
		__syncthreads();
	}

	// write out the results
	for (uint resIdxM = 0; resIdxM < TM; resIdxM += 1) {
		for (uint resIdxN = 0; resIdxN < TN; resIdxN += 4) {
			// load C vector into registers
			float4 tmp = reinterpret_cast<float4*>(
				&C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN])[0];
			// perform GEMM update in reg
			tmp.x = threadResults[resIdxM * TN + resIdxN] + tmp.x;
			tmp.y = threadResults[resIdxM * TN + resIdxN + 1] + tmp.y;
			tmp.z = threadResults[resIdxM * TN + resIdxN + 2] + tmp.z;
			tmp.w = threadResults[resIdxM * TN + resIdxN + 3] + tmp.w;
			// write back
			reinterpret_cast<float4*>(
				&C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN])[0] =
				tmp;
		}
	}
}

void runSgemmVectorize(int N, float* A, float* B, float* C) {
	const int BK = 8;
	const int TM = 8;
	const int TN = 8;

	const int BM = 128;
	const int BN = 128;
	dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(N, BM));
	dim3 blockDim((BM * BN) / (TM * TN));
	sgemmVectorize<BM, BN, BK, TM, TN>
		<< <gridDim, blockDim >> > (N, A, B, C);

}

/* =========================================================================== */

// Check result on the CPU
void verify_result(vector<float>& a, vector<float>& b, vector<float>& c, int N) {
	// For every row...
	for (int i = 0; i < N; i++) {
		// For every column...
		for (int j = 0; j < N; j++) {
			// For every element in the row-column pair
			int tmp = 0;
			for (int k = 0; k < N; k++) {
				// Accumulate the partial results
				tmp += a[i * N + k] * b[k * N + j];
			}

			// Check against the CPU result
			assert(tmp == c[i * N + j]);
		}
	}
}

void verify_resultv1(std::vector<float>& a, std::vector<float>& b, std::vector<float>& c, int N) {
	const float epsilon = 1e-3f;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			float tmp = 0.0f;
			for (int k = 0; k < N; k++) {
				tmp += a[i * N + k] * b[k * N + j];
			}

			if (std::abs(tmp - c[i * N + j]) > epsilon) {
				printf("Mismatch at (%d, %d): expected %.5f, got %.5f\n", i, j, tmp, c[i * N + j]);
				exit(1); // or return; or break;
			}
		}
	}
	printf("All results matched!\n");
}

int main() {
	// Matrix size of N x N;
	int N = 8192;

	// Size (in bytes) of matrix
	size_t bytes = N * N * sizeof(float);

	//Nbr of Floating Operations
	float Nbr_GFLOPS;
	Nbr_GFLOPS = 2 * N / 1000.0 * N / 1000.0 * N / 1000.0;

	// Host vectors
	vector<float> h_a(N * N);
	vector<float> h_b(N * N);
	vector<float> h_c(N * N);

	cout << "Step1 : h_a and h_b generation \n";

	// Initialize matrices
	generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
	generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

	cout << "Step2 : Mem Allocation on host \n";
	// Allocate device memory
	float* d_a, * d_b;
	float* d_c;
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	cout << "Step3 : Launch Event to measure Time \n";
	// --- start to count execution time of GPU version ---
	float Total_gpu_time, Host2Dev_time, Kernel_time, Dev2Host_time;
	// some events to count the execution time
	hipEvent_t start, stop, Host2dev, KernelExec;

	hipEventCreate(&start);
	hipEventCreate(&Host2dev);
	hipEventCreate(&KernelExec);
	hipEventCreate(&stop);
	// --- execution time of GPU version ---

	hipEventRecord(start, 0);


	// Copy data to the device
	cout << "Step3 : Copy Data To Device \n";
	hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

	hipEventRecord(Host2dev, 0);

	// Launch kernel
	runSgemmVectorize(N, d_a, d_b, d_c);

	// record time after kernel execution
	hipEventRecord(KernelExec, 0);


	// Copy back to the host
	hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);


	hipDeviceSynchronize();
	// time counting terminate
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// compute time elapse on GPU computing
	hipEventElapsedTime(&Total_gpu_time, start, stop);
	hipEventElapsedTime(&Host2Dev_time, start, Host2dev);
	hipEventElapsedTime(&Kernel_time, Host2dev, KernelExec);
	hipEventElapsedTime(&Dev2Host_time, KernelExec, stop);



	printf("Time elapsed on Host To Device Transfer: %f ms.\n\n", Host2Dev_time);
	printf("Time elapsed on matrix multiplication on GPU: %f ms.\n\n", Kernel_time);
	printf("Time elapsed on Device To Host Transfer: %f ms.\n\n", Dev2Host_time);
	printf("Total Time: %f ms.\n\n", Total_gpu_time);


	float Perf_GFLOPS;
	Perf_GFLOPS = Nbr_GFLOPS * 1000 / Kernel_time;
	printf("Kernel Execution Performance: %f GFLOPS.\n\n", Perf_GFLOPS);


	// Check result
	verify_result(h_a, h_b, h_c, N);

	cout << "COMPLETED SUCCESSFULLY\n";

	// Free memory on device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	//wait for keyboard press
	int kml;
	scanf("%c", &kml);

	return 0;
}
